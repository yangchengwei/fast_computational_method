#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#define _USE_MATH_DEFINES

#include <stdio.h>
#include <stdlib.h> 
#include <math.h>
#include <time.h>

#define THREADS_PER_BLOCK 512

#define PRINT_RESULT 1
#define DEBUG_OTHER 0
#define DEBUG 0

/* constant */

__constant__ double w3_im2;
__constant__ double w5_re1;
__constant__ double w5_re2;
__constant__ double w5_im3;
__constant__ double w5_im4;
__constant__ double w7_re1;
__constant__ double w7_im1;
__constant__ double w7_re2;
__constant__ double w7_im2;
__constant__ double w7_re3;
__constant__ double w7_im3;

/* functions declaration */

hipError_t fftCuda(double *x_re, double *x_im, int N);

/* kernel */

__global__ void butterflyKernel_7(double *x_re, double *x_im, double *w_re, double *w_im, const int m, const int s, const int M)
{
	int k, A, B, C, D, E, F, G;
	double tA_re, tA_im, tB_re, tB_im, tw_re, tw_im, tC_re, tC_im;
	double tD_re, tD_im, tE_re, tE_im, tF_re, tF_im, tG_re, tG_im, t;
	
	if ((blockIdx.x * THREADS_PER_BLOCK + threadIdx.x) >= M) return;
	
	k = (blockIdx.x * THREADS_PER_BLOCK + threadIdx.x) % m;
	A = ((blockIdx.x * THREADS_PER_BLOCK + threadIdx.x) / m) * s + k;
	B = A + m;
	C = B + m;
	D = C + m;
	E = D + m;
	F = E + m;
	G = F + m;

	tA_re = x_re[A];
	tA_im = x_im[A];
	tB_re = w_re[k] * x_re[B] - w_im[k] * x_im[B];
	tB_im = w_re[k] * x_im[B] + w_im[k] * x_re[B];
	tw_re = w_re[k] * w_re[k] - w_im[k] * w_im[k];
	tw_im = 2 * w_re[k] * w_im[k];
	tC_re = tw_re*x_re[C] - tw_im*x_im[C];
	tC_im = tw_re*x_im[C] + tw_im*x_re[C];
	t = tw_re;
	tw_re = tw_re*w_re[k] - tw_im*w_im[k];
	tw_im = t    *w_im[k] + tw_im*w_re[k];
	tD_re = tw_re*x_re[D] - tw_im*x_im[D];
	tD_im = tw_re*x_im[D] + tw_im*x_re[D];
	t = tw_re;
	tw_re = tw_re*w_re[k] - tw_im*w_im[k];
	tw_im = t    *w_im[k] + tw_im*w_re[k];
	tE_re = tw_re*x_re[E] - tw_im*x_im[E];
	tE_im = tw_re*x_im[E] + tw_im*x_re[E];
	t = tw_re;
	tw_re = tw_re*w_re[k] - tw_im*w_im[k];
	tw_im = t    *w_im[k] + tw_im*w_re[k];
	tF_re = tw_re*x_re[F] - tw_im*x_im[F];
	tF_im = tw_re*x_im[F] + tw_im*x_re[F];
	t = tw_re;
	tw_re = tw_re*w_re[k] - tw_im*w_im[k];
	tw_im = t    *w_im[k] + tw_im*w_re[k];
	tG_re = tw_re*x_re[G] - tw_im*x_im[G];
	tG_im = tw_re*x_im[G] + tw_im*x_re[G];
	
	x_re[A] = tA_re + tB_re + tC_re + tD_re + tE_re + tF_re + tG_re;
	x_re[B] = tA_re + (tB_re + tG_re)*w7_re1 + (tG_im - tB_im)*w7_im1 + (tC_re + tF_re)*w7_re2 + (tF_im - tC_im)*w7_im2 + (tD_re + tE_re)*w7_re3 + (tE_im - tD_im)*w7_im3;
	x_re[C] = tA_re + (tB_re + tG_re)*w7_re2 + (tG_im - tB_im)*w7_im2 + (tC_re + tF_re)*w7_re3 + (tC_im - tF_im)*w7_im3 + (tD_re + tE_re)*w7_re1 + (tD_im - tE_im)*w7_im1;
	x_re[D] = tA_re + (tB_re + tG_re)*w7_re3 + (tG_im - tB_im)*w7_im3 + (tC_re + tF_re)*w7_re1 + (tC_im - tF_im)*w7_im1 + (tD_re + tE_re)*w7_re2 + (tE_im - tD_im)*w7_im2;
	x_re[E] = tA_re + (tB_re + tG_re)*w7_re3 + (tB_im - tG_im)*w7_im3 + (tC_re + tF_re)*w7_re1 + (tF_im - tC_im)*w7_im1 + (tD_re + tE_re)*w7_re2 + (tD_im - tE_im)*w7_im2;
	x_re[F] = tA_re + (tB_re + tG_re)*w7_re2 + (tB_im - tG_im)*w7_im2 + (tC_re + tF_re)*w7_re3 + (tF_im - tC_im)*w7_im3 + (tD_re + tE_re)*w7_re1 + (tE_im - tD_im)*w7_im1;
	x_re[G] = tA_re + (tB_re + tG_re)*w7_re1 + (tB_im - tG_im)*w7_im1 + (tC_re + tF_re)*w7_re2 + (tC_im - tF_im)*w7_im2 + (tD_re + tE_re)*w7_re3 + (tD_im - tE_im)*w7_im3;
	x_im[A] = tA_im + tB_im + tC_im + tD_im + tE_im + tF_im + tG_im;
	x_im[B] = tA_im + (tB_im + tG_im)*w7_re1 + (tB_re - tG_re)*w7_im1 + (tC_im + tF_im)*w7_re2 + (tC_re - tF_re)*w7_im2 + (tD_im + tE_im)*w7_re3 + (tD_re - tE_re)*w7_im3;
	x_im[C] = tA_im + (tB_im + tG_im)*w7_re2 + (tB_re - tG_re)*w7_im2 + (tC_im + tF_im)*w7_re3 + (tF_re - tC_re)*w7_im3 + (tD_im + tE_im)*w7_re1 + (tE_re - tD_re)*w7_im1;
	x_im[D] = tA_im + (tB_im + tG_im)*w7_re3 + (tB_re - tG_re)*w7_im3 + (tC_im + tF_im)*w7_re1 + (tF_re - tC_re)*w7_im1 + (tD_im + tE_im)*w7_re2 + (tD_re - tE_re)*w7_im2;
	x_im[E] = tA_im + (tB_im + tG_im)*w7_re3 + (tG_re - tB_re)*w7_im3 + (tC_im + tF_im)*w7_re1 + (tC_re - tF_re)*w7_im1 + (tD_im + tE_im)*w7_re2 + (tE_re - tD_re)*w7_im2;
	x_im[F] = tA_im + (tB_im + tG_im)*w7_re2 + (tG_re - tB_re)*w7_im2 + (tC_im + tF_im)*w7_re3 + (tC_re - tF_re)*w7_im3 + (tD_im + tE_im)*w7_re1 + (tD_re - tE_re)*w7_im1;
	x_im[G] = tA_im + (tB_im + tG_im)*w7_re1 + (tG_re - tB_re)*w7_im1 + (tC_im + tF_im)*w7_re2 + (tF_re - tC_re)*w7_im2 + (tD_im + tE_im)*w7_re3 + (tE_re - tD_re)*w7_im3;
}
__global__ void butterflyKernel_5(double *x_re, double *x_im, double *w_re, double *w_im, const int m, const int s, const int M)
{
	int k, A, B, C, D, E;
	double tA_re, tA_im, tB_re, tB_im, tw_re, tw_im, tC_re, tC_im;
	double tD_re, tD_im, tE_re, tE_im, t;

	if ((blockIdx.x * THREADS_PER_BLOCK + threadIdx.x) >= M) return;

	k = (blockIdx.x * THREADS_PER_BLOCK + threadIdx.x) % m;
	A = ((blockIdx.x * THREADS_PER_BLOCK + threadIdx.x) / m) * s + k;
	B = A + m;
	C = B + m;
	D = C + m;
	E = D + m;

	tA_re = x_re[A];
	tA_im = x_im[A];
	tB_re = w_re[k] * x_re[B] - w_im[k] * x_im[B];
	tB_im = w_re[k] * x_im[B] + w_im[k] * x_re[B];
	tw_re = w_re[k] * w_re[k] - w_im[k] * w_im[k];
	tw_im = 2 * w_re[k] * w_im[k];
	tC_re = tw_re*x_re[C] - tw_im*x_im[C];
	tC_im = tw_re*x_im[C] + tw_im*x_re[C];
	t = tw_re;
	tw_re = tw_re*w_re[k] - tw_im*w_im[k];
	tw_im = t    *w_im[k] + tw_im*w_re[k];
	tD_re = tw_re*x_re[D] - tw_im*x_im[D];
	tD_im = tw_re*x_im[D] + tw_im*x_re[D];
	t = tw_re;
	tw_re = tw_re*w_re[k] - tw_im*w_im[k];
	tw_im = t    *w_im[k] + tw_im*w_re[k];
	tE_re = tw_re*x_re[E] - tw_im*x_im[E];
	tE_im = tw_re*x_im[E] + tw_im*x_re[E];

	x_re[A] = tA_re + tB_re + tC_re + tD_re + tE_re;
	x_re[B] = tA_re + w5_re1*(tB_re + tE_re) + w5_im4*(tB_im - tE_im) + w5_re2*(tC_re + tD_re) + w5_im3*(tC_im - tD_im);
	x_re[C] = tA_re + w5_re2*(tB_re + tE_re) + w5_im3*(tB_im - tE_im) + w5_re1*(tC_re + tD_re) + w5_im4*(tD_im - tC_im);
	x_re[D] = tA_re + w5_re2*(tB_re + tE_re) + w5_im3*(tE_im - tB_im) + w5_re1*(tC_re + tD_re) + w5_im4*(tC_im - tD_im);
	x_re[E] = tA_re + w5_re1*(tB_re + tE_re) + w5_im4*(tE_im - tB_im) + w5_re2*(tC_re + tD_re) + w5_im3*(tD_im - tC_im);
	x_im[A] = tA_im + tB_im + tC_im + tD_im + tE_im;
	x_im[B] = tA_im + w5_re1*(tB_im + tE_im) + w5_im4*(tE_re - tB_re) + w5_re2*(tC_im + tD_im) + w5_im3*(tD_re - tC_re);
	x_im[C] = tA_im + w5_re2*(tB_im + tE_im) + w5_im3*(tE_re - tB_re) + w5_re1*(tC_im + tD_im) + w5_im4*(tC_re - tD_re);
	x_im[D] = tA_im + w5_re2*(tB_im + tE_im) + w5_im3*(tB_re - tE_re) + w5_re1*(tC_im + tD_im) + w5_im4*(tD_re - tC_re);
	x_im[E] = tA_im + w5_re1*(tB_im + tE_im) + w5_im4*(tB_re - tE_re) + w5_re2*(tC_im + tD_im) + w5_im3*(tC_re - tD_re);
}
__global__ void butterflyKernel_3(double *x_re, double *x_im, double *w_re, double *w_im, const int m, const int s, const int M)
{
	int k, A, B, C;
	double tA_re, tA_im, tB_re, tB_im, tw_re, tw_im, tC_re, tC_im;

	if ((blockIdx.x * THREADS_PER_BLOCK + threadIdx.x) >= M) return;

	k = (blockIdx.x * THREADS_PER_BLOCK + threadIdx.x) % m;
	A = ((blockIdx.x * THREADS_PER_BLOCK + threadIdx.x) / m) * s + k;
	B = A + m;
	C = B + m;

	tA_re = x_re[A];
	tA_im = x_im[A];
	tB_re = w_re[k] * x_re[B] - w_im[k] * x_im[B];
	tB_im = w_re[k] * x_im[B] + w_im[k] * x_re[B];
	tw_re = w_re[k] * w_re[k] - w_im[k] * w_im[k];
	tw_im = 2 * w_re[k] * w_im[k];
	tC_re = tw_re*x_re[C] - tw_im*x_im[C];
	tC_im = tw_re*x_im[C] + tw_im*x_re[C];

	x_re[A] = tA_re + tB_re + tC_re;
	x_im[A] = tA_im + tB_im + tC_im;
	x_re[B] = tA_re - 0.5*(tB_re + tC_re) + w3_im2*(tB_im - tC_im);
	x_im[B] = tA_im + w3_im2*(tC_re - tB_re) - 0.5*(tB_im + tC_im);
	x_re[C] = tA_re - 0.5*(tB_re + tC_re) + w3_im2*(tC_im - tB_im);
	x_im[C] = tA_im + w3_im2*(tB_re - tC_re) - 0.5*(tB_im + tC_im);
}
__global__ void butterflyKernel_2(double *x_re, double *x_im, double *w_re, double *w_im, const int m, const int s, const int M)
{
	int k, A, B;
	double tA_re, tA_im, tB_re, tB_im;

	if ((blockIdx.x * THREADS_PER_BLOCK + threadIdx.x) >= M) return;

	k = (blockIdx.x * THREADS_PER_BLOCK + threadIdx.x) % m;
	A = ((blockIdx.x * THREADS_PER_BLOCK + threadIdx.x) / m) * s + k;
	B = A + m;

	tA_re = x_re[A];
	tA_im = x_im[A];
	tB_re = w_re[k] * x_re[B] - w_im[k] * x_im[B];
	tB_im = w_re[k] * x_im[B] + w_im[k] * x_re[B];

	x_re[A] = tA_re + tB_re;
	x_re[B] = tA_re - tB_re;
	x_im[A] = tA_im + tB_im;
	x_im[B] = tA_im - tB_im;
}

/* main */

int main()
{
	int i, N;
	double *x_re, *x_im, t;
	clock_t t1, t2;
	hipError_t cudaStatus;

	printf("N=");
	scanf("%d", &N);
	printf("N=%d\n", N);

	x_re = (double *)malloc(N * sizeof(double));
	x_im = (double *)malloc(N * sizeof(double));

	/* initial CUDA */

	// Choose which GPU to run on.
	hipFree(0);
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) { printf("cudaNotSuccess\n"); system("pause"); exit(cudaStatus); }

	// constant 
	t = sqrt(3.0) / 2.0;	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(w3_im2), &t, 8);
	if (cudaStatus != hipSuccess) { printf("cudaNotSuccess w3_im2\n"); system("pause"); exit(cudaStatus); }
	t = cos(2.0*M_PI / 5.0);	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(w5_re1), &t, 8);
	if (cudaStatus != hipSuccess) { printf("cudaNotSuccess w5_re1\n"); system("pause"); exit(cudaStatus); }
	t = cos(4.0*M_PI / 5.0);	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(w5_re2), &t, 8);
	if (cudaStatus != hipSuccess) { printf("cudaNotSuccess w5_re1\n"); system("pause"); exit(cudaStatus); }
	t = sin(4.0*M_PI / 5.0);	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(w5_im3), &t, 8);
	if (cudaStatus != hipSuccess) { printf("cudaNotSuccess w5_re1\n"); system("pause"); exit(cudaStatus); }
	t = sin(2.0*M_PI / 5.0);	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(w5_im4), &t, 8);
	if (cudaStatus != hipSuccess) { printf("cudaNotSuccess w5_re1\n"); system("pause"); exit(cudaStatus); }
	t = cos(2.0*M_PI / 7.0);	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(w7_re1), &t, 8);
	if (cudaStatus != hipSuccess) { printf("cudaNotSuccess w5_re1\n"); system("pause"); exit(cudaStatus); }
	t = -sin(2.0*M_PI / 7.0);	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(w7_im1), &t, 8);
	if (cudaStatus != hipSuccess) { printf("cudaNotSuccess w5_re1\n"); system("pause"); exit(cudaStatus); }
	t = cos(4.0*M_PI / 7.0);	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(w7_re2), &t, 8);
	if (cudaStatus != hipSuccess) { printf("cudaNotSuccess w5_re1\n"); system("pause"); exit(cudaStatus); }
	t = -sin(4.0*M_PI / 7.0);	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(w7_im2), &t, 8);
	if (cudaStatus != hipSuccess) { printf("cudaNotSuccess w5_re1\n"); system("pause"); exit(cudaStatus); }
	t = cos(6.0*M_PI / 7.0);	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(w7_re3), &t, 8);
	if (cudaStatus != hipSuccess) { printf("cudaNotSuccess w5_re1\n"); system("pause"); exit(cudaStatus); }
	t = -sin(6.0*M_PI / 7.0);	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(w7_im3), &t, 8);
	if (cudaStatus != hipSuccess) { printf("cudaNotSuccess w5_re1\n"); system("pause"); exit(cudaStatus); }

	/* main */

	for (i = 0; i<N; ++i)
	{
		x_re[i] = i;
		x_im[i] = 0.0;
	}

	t1 = clock();

	fftCuda(x_re, x_im, N);

	t2 = clock();

	printf("time = %f\n", 1.0*(t2 - t1) / (double)CLOCKS_PER_SEC);

#if PRINT_RESULT
	system("pause");
	for (i = 0; i<N; i++)
	{
		printf("%f + %f i\n", x_re[i], x_im[i]);
	}
#endif

	/* other */

#if DEBUG_OTHER 

	for (i = 0; i<N; ++i)
	{
		x_re[i] = i;
		x_im[i] = 0.0;
	}

	t1 = clock();

	//fftHost(x_re, x_im, N);

	t2 = clock();

	printf("time = %f\n", 1.0*(t2 - t1) / (double)CLOCKS_PER_SEC);

#if PRINT_RESULT
	system("pause");
	for (i = 0; i<N; i++)
	{
		printf("%f + %f i\n", x_re[i], x_im[i]);
	}
#endif

#endif

	system("pause");
	return 0;
}

/* functions definition */

hipError_t fftCuda(double *x_re, double *x_im, int N)
{
	/* initial */

	// Device variable
	int memorySize = N * sizeof(double);
	double *dev_x_re, *dev_x_im;
	double *dev_w_re, *dev_w_im;
	hipError_t cudaStatus;

	// Device memory
	cudaStatus = hipMalloc((void**)&dev_x_re, memorySize);
	if (cudaStatus != hipSuccess) { printf("cudaNotSuccess\n"); system("pause"); exit(cudaStatus); }
	cudaStatus = hipMalloc((void**)&dev_x_im, memorySize);
	if (cudaStatus != hipSuccess) { printf("cudaNotSuccess\n"); system("pause"); exit(cudaStatus); }
	cudaStatus = hipMalloc((void**)&dev_w_re, memorySize);
	if (cudaStatus != hipSuccess) { printf("cudaNotSuccess: omega allocate %s\n", hipGetErrorString(cudaStatus)); system("pause"); exit(cudaStatus); }
	cudaStatus = hipMalloc((void**)&dev_w_im, memorySize);
	if (cudaStatus != hipSuccess) { printf("cudaNotSuccess: omega allocate %s\n", hipGetErrorString(cudaStatus)); system("pause"); exit(cudaStatus); }

	// Host variable
	int copy_N = N, power_2 = 0, power_3 = 0, power_5 = 0, power_7 = 0, power_75 = 0, power_753 = 0, power_sum = 0;
	int p, q, i, k, m, s, M, step, gate, add, *order;
	double w_N_re, w_N_im, t, *temp_re, *temp_im;

	// Power Computation
	while (copy_N % 7 == 0) { power_7++; copy_N /= 7; }
	while (copy_N % 5 == 0) { power_5++; copy_N /= 5; }
	while (copy_N % 3 == 0) { power_3++; copy_N /= 3; }
	while (copy_N % 2 == 0) { power_2++; copy_N /= 2; }
	if (copy_N != 1) { printf("ERROR: N is not radix-2,3,5,7 !\n"); system("pause"); exit(EXIT_FAILURE);}
	power_sum = power_7 + power_5 + power_3 + power_2;
	power_753 = power_7 + power_5 + power_3;
	power_75 = power_7 + power_5;

	// Host memory
	temp_re = (double *)malloc(N * sizeof(double));
	if (temp_re == NULL) { printf("Failed to allocate host memory temp_re!\n"); system("pause"); exit(EXIT_FAILURE); }
	temp_im = (double *)malloc(N * sizeof(double));
	if (temp_im == NULL) { printf("Failed to allocate host memory temp_im!\n"); system("pause"); exit(EXIT_FAILURE); }
	order = (int *)malloc(power_sum * sizeof(int));
	if (order == NULL) { printf("Failed to allocate host memory order!\n"); system("pause"); exit(EXIT_FAILURE); }

	// order
	for (i = 0; i<power_7; i++)				order[i] = 7;
	for (i = power_7; i<power_75; i++)		order[i] = 5;
	for (i = power_75; i<power_753; i++)	order[i] = 3;
	for (i = power_753; i<power_sum; i++)	order[i] = 2;



	/* FFT */

	/* bit reverse */

	// copy x
	for (i = 0; i<N; i++)
	{
		temp_re[i] = x_re[i];
		temp_im[i] = x_im[i];
	}

#if DEBUG
	clock_t T1, T2;
	T1 = clock();
#endif

	// bit reverse main
	step = N / order[0];
	q = step;			// first change
	for (p = 1; p<N - 1; p++)
	{
		// change value
		x_re[p] = temp_re[q];
		x_im[p] = temp_im[q];

		// compute next place
		i = 0;
		add = step;
		gate = (order[i++] - 1)*add;
		while (q >= gate && gate > 0)
		{
			q = q - gate;
			add = add / order[i];
			gate = (order[i++] - 1)*add;
		}
		q = q + add;
	}

#if DEBUG
	T2 = clock();
	printf("bit reverse time = %f\n", 1.0*(T2 - T1) / (double)CLOCKS_PER_SEC);
#endif

	// length N array copy
	cudaStatus = hipMemcpy(dev_x_re, x_re, memorySize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("cudaNotSuccess\n"); system("pause"); exit(cudaStatus); }
	cudaStatus = hipMemcpy(dev_x_im, x_im, memorySize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("cudaNotSuccess\n"); system("pause"); exit(cudaStatus); }



	/* butterfly */

	m = 1;
	s = 1;
	// parallel
	for (i = 0; i<power_7; i++)
	{
		s *= order[i];

		// omega computation
		temp_re[0] = 1.0;
		temp_im[0] = 0.0;
		w_N_re = cos(2.0*M_PI / s);
		w_N_im = -sin(2.0*M_PI / s);
		for (k = 1; k<m; ++k)
		{
			temp_re[k] = w_N_re*temp_re[k - 1] - w_N_im*temp_im[k - 1];
			temp_im[k] = w_N_re*temp_im[k - 1] + w_N_im*temp_re[k - 1];
		}

		// omega copy
		cudaStatus = hipMemcpy(dev_w_re, temp_re, m * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) { printf("cudaNotSuccess: omega copy\n"); system("pause"); exit(cudaStatus); }
		cudaStatus = hipMemcpy(dev_w_im, temp_im, m * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) { printf("cudaNotSuccess: omega copy\n"); system("pause"); exit(cudaStatus); }

		// kernel
		M = N / order[i];
#if DEBUG
		printf("blockSize, threadSize: %d, %d\n", M / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK);
#endif
		butterflyKernel_7 << < M / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK >> >
			(dev_x_re, dev_x_im, dev_w_re, dev_w_im, m, s, M);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) { printf("cudaNotSuccess: kernel\n"); system("pause"); exit(cudaStatus); }

		m *= order[i];
	}
	for (i = power_7; i<power_75; i++)
	{
		s *= order[i];

		// omega computation
		temp_re[0] = 1.0;
		temp_im[0] = 0.0;
		w_N_re = cos(2.0*M_PI / s);
		w_N_im = -sin(2.0*M_PI / s);
		for (k = 1; k<m; ++k)
		{
			temp_re[k] = w_N_re*temp_re[k - 1] - w_N_im*temp_im[k - 1];
			temp_im[k] = w_N_re*temp_im[k - 1] + w_N_im*temp_re[k - 1];
		}

		// omega copy
		cudaStatus = hipMemcpy(dev_w_re, temp_re, m * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) { printf("cudaNotSuccess: omega copy\n"); system("pause"); exit(cudaStatus); }
		cudaStatus = hipMemcpy(dev_w_im, temp_im, m * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) { printf("cudaNotSuccess: omega copy\n"); system("pause"); exit(cudaStatus); }

		// kernel
		M = N / order[i];
#if DEBUG
		printf("blockSize, threadSize: %d, %d\n", M / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK);
#endif
		butterflyKernel_5 << < M / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK >> >
			(dev_x_re, dev_x_im, dev_w_re, dev_w_im, m, s, M);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) { printf("cudaNotSuccess: kernel\n"); system("pause"); exit(cudaStatus); }

		m *= order[i];
	}
	for (i = power_75; i<power_753; i++)
	{
		s *= order[i];

		// omega computation
		temp_re[0] = 1.0;
		temp_im[0] = 0.0;
		w_N_re = cos(2.0*M_PI / s);
		w_N_im = -sin(2.0*M_PI / s);
		for (k = 1; k<m; ++k)
		{
			temp_re[k] = w_N_re*temp_re[k - 1] - w_N_im*temp_im[k - 1];
			temp_im[k] = w_N_re*temp_im[k - 1] + w_N_im*temp_re[k - 1];
		}

		// omega copy
		cudaStatus = hipMemcpy(dev_w_re, temp_re, m * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) { printf("cudaNotSuccess: omega copy\n"); system("pause"); exit(cudaStatus); }
		cudaStatus = hipMemcpy(dev_w_im, temp_im, m * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) { printf("cudaNotSuccess: omega copy\n"); system("pause"); exit(cudaStatus); }

		// kernel
		M = N / order[i];
#if DEBUG
		printf("blockSize, threadSize: %d, %d\n", M / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK);
#endif
		butterflyKernel_3 << < M / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK >> >
			(dev_x_re, dev_x_im, dev_w_re, dev_w_im, m, s, M);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) { printf("cudaNotSuccess: kernel\n"); system("pause"); exit(cudaStatus); }

		m *= order[i];
	}
	for (i = power_753; i<power_sum; i++)
	{
		s *= order[i];

		// omega computation
		temp_re[0] = 1.0;
		temp_im[0] = 0.0;
		w_N_re = cos(2.0*M_PI / s);
		w_N_im = -sin(2.0*M_PI / s);
		for (k = 1; k<m; ++k)
		{
			temp_re[k] = w_N_re*temp_re[k - 1] - w_N_im*temp_im[k - 1];
			temp_im[k] = w_N_re*temp_im[k - 1] + w_N_im*temp_re[k - 1];
		}

		// omega copy
		cudaStatus = hipMemcpy(dev_w_re, temp_re, m * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) { printf("cudaNotSuccess: omega copy\n"); system("pause"); exit(cudaStatus); }
		cudaStatus = hipMemcpy(dev_w_im, temp_im, m * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) { printf("cudaNotSuccess: omega copy\n"); system("pause"); exit(cudaStatus); }

		// kernel
		M = N / order[i];
#if DEBUG
		printf("blockSize, threadSize: %d, %d\n", M / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK);
#endif
		butterflyKernel_2 << < M / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK >> >
			(dev_x_re, dev_x_im, dev_w_re, dev_w_im, m, s, M);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) { printf("cudaNotSuccess: kernel\n"); system("pause"); exit(cudaStatus); }

		m *= order[i];
	}

	/* FINISH */

	// Memory
	cudaStatus = hipMemcpy(x_re, dev_x_re, memorySize, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { printf("cudaNotSuccess\n"); system("pause"); exit(cudaStatus); }
	cudaStatus = hipMemcpy(x_im, dev_x_im, memorySize, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { printf("cudaNotSuccess\n"); system("pause"); exit(cudaStatus); }

	return cudaStatus;
}